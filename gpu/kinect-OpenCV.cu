#include "hip/hip_runtime.h"
/*
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <stdio.h>
#include <time.h>
#include "libfreenect/libfreenect.h"
#include <pthread.h>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "structures.h"

pthread_t freenect_thread;
int terminate = 0;
uint8_t *rgb_display;
uint8_t *gray_display;
uint8_t *tmp,*tmp2;
freenect_context *f_ctx;
freenect_device *f_dev;
uint8_t* window;

/*
*	Grayscale conversion
*/

void gray(uint8_t *frame){ //Grayscale Conversion Function
	int row,col,layer;
	for(row=0;row<HEIGHT;row++){
		for(col=0;col<WIDTH;col++){
			for(layer=0;layer<3;layer++)
				window[layer]=frame[row*(WIDTH*3)+col*3+layer];
			tmp[row*WIDTH+col]=(window[0]+window[1]+window[2])/3;
		}
	}
}

/*
*	CUDA Section
*/

__device__ void sort(uint8_t* window){ // Sort Function for Device
	uint8_t temp;
	int j,i;
	for(i=1;i<9;i++){
		for(j=0;j<9-i;j++){
			if(window[j] > window[j+1]){
				temp=window[j];
				window[j]=window[j+1];
				window[j+1]=temp;
			}
		}
	}
}

__device__ void medianFilter(uint8_t *frame){ //Median Filter
	int blocknum = blockIdx.x;
	int framestart = blocknum*(SIZE/CORENUM);
	int rowcount=(HEIGHT/CORENUM-1);
	int row,col,wrow,wcol,s,flag=0,received_row;
	uint8_t temp,tmp[WIDTH];
	uint8_t window[9];
	/* Get number of rows received according to block number */
	if(blocknum==CORENUM-1 || blocknum==0){
		received_row=HEIGHT/CORENUM+1;
	}
	else{
		received_row=HEIGHT/CORENUM+2;
	}
	if(blocknum==0){
		for(row=0;row<received_row-1;row++){
			for(col=0;col<WIDTH;col++){
				//Fill Window
				for(wrow=-1;wrow<2;wrow++){
					if(wrow==-1 && row==0){
						wrow=0;
						flag=1;
					}
					for(wcol=-1;wcol<2;wcol++){
						if(col==0){
							if(wcol==-1){
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+(wcol+1)];
							}
							else{
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
							}
						}
						else if(col==WIDTH-1){
							if(wcol==2){
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+(wcol-1)];
							}
							else{
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
							}
						}
						else{
							window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
						}
					}
					if(flag){
						flag=0;
						wrow=-1;
					}
				}
				//Sort Window
				sort(window);
				s=1;
				if (window[4]==(uint8_t) 0){
					while(window[4+s]==(uint8_t) 0 && s!=5){
						s=s+1;
					}
					window[4]=window[4+s+(10-(5+s))/2];
				}
				if (window[4]==(uint8_t) 255){
					while(window[4-s]==(uint8_t) 255 && s!=5){
						s=s+1;
					}
					window[4]=window[(4-s)/2];
				}
				if(row==0){
					tmp[col]=window[4];
				}
				else{// Fix to not use changed bit in next iteration
					if(col==0){
						temp=tmp[col];
						tmp[col]=window[4];
					}
					else if(col==WIDTH-1){
						frame[framestart+(row-1)*WIDTH+col-1]=temp;
						frame[framestart+(row-1)*WIDTH+col]=tmp[col];
						tmp[col]=window[4];
					}
					else{
						frame[framestart+(row-1)*WIDTH+col-1]=temp;
						temp=tmp[col];
						tmp[col]=window[4];
					}
				}
			}
		}
	}
	else if(blocknum==CORENUM-1){
		for(row=1;row<received_row;row++){
			for(col=0;col<WIDTH;col++){
				//Fill Window
				for(wrow=-1;wrow<2;wrow++){
					if(wrow==1 && row==received_row-1){
						wrow=0;
						flag=1;
					}
					for(wcol=-1;wcol<2;wcol++){
						if(col==0){
							if(wcol==-1){
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+(wcol+1)];
							}
							else{
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
							}
						}
						else if(col==WIDTH-1){
							if(wcol==2){
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+(wcol-1)];
							}
							else{
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
							}
						}
						else{
							window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
						}
					}
					if(flag){
						flag=0;
						break;
					}
				}
				//Sort Window
				sort(window);
				s=1;
				if (window[4]==(uint8_t) 0){
					while(window[4+s]==(uint8_t) 0 && s!=5){
						s=s+1;
					}
					window[4]=window[4+s+(10-(5+s))/2];
				}
				if (window[4]==(uint8_t) 255){
					while(window[4-s]==(uint8_t) 255 && s!=5){
						s=s+1;
					}
					window[4]=window[(4-s)/2];
				}
				if(row==1){
					tmp[col]=window[4];
				}
				else{
					frame[framestart+(row-2)*WIDTH+col]=tmp[col];
					tmp[col]=window[4];
				}
			}
		}
	}
	else{
		for(row=1;row<received_row-1;row++){
			for(col=0;col<WIDTH;col++){
				//Fill Window
				for(wrow=-1;wrow<2;wrow++){
					for(wcol=-1;wcol<2;wcol++){
						if(col==0){
							if(wcol==-1){
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+(wcol+1)];
							}
							else{
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
							}
						}
						else if(col==WIDTH-1){
							if(wcol==2){
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+(wcol-1)];
							}
							else{
								window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
							}
						}
						else{
							window[(wrow+1)*3+(wcol+1)]=frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
						}
					}
				}
				//Sort Window
				sort(window);
				s=1;
				if (window[4]==(uint8_t) 0){
					while(window[4+s]==(uint8_t) 0 && s!=5){
						s=s+1;
					}
					window[4]=window[4+s+(10-(5+s))/2];
				}
				if (window[4]==(uint8_t) 255){
					while(window[4-s]==(uint8_t) 255 && s!=5){
						s=s+1;
					}
					window[4]=window[(4-s)/2];
				}
				if(row==1){
					tmp[col]=window[4];
				}
				else{
					frame[framestart+(row-2)*WIDTH+col]=tmp[col];
					tmp[col]=window[4];
				}
			}
		}
	}
	for (col=0;col<WIDTH;col++){ // Final row written from tmp to frame
		frame[framestart+WIDTH*rowcount+col]=tmp[col];
	}
}

__device__ void edgeDetection(uint8_t *frame){ // Laplacian Filter as an Edge Detector
	int blocknum = blockIdx.x;
	int rowcount=(HEIGHT/CORENUM-1);
	int framestart = blocknum*(SIZE/CORENUM);
	int row=0,col,value,wrow,wcol;
	int window[9] = {-1,-1,-1,-1,8,-1,-1,-1,-1}; // Laplacian Kernel
	uint8_t temp,tmp[WIDTH];
	for(col=0;col<WIDTH;col++){ //row 0
		if(col==0){
			value=0;
			for(wrow=0;wrow<2;wrow++){
				for(wcol=0;wcol<2;wcol++){
					value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
				}
			}
			tmp[col]=(uint8_t) (value/4<0 ? 0 : 255);
		}
		else if(col==WIDTH-1){
			value=0;
			for(wrow=0;wrow<2;wrow++){
				for(wcol=-1;wcol<1;wcol++){
					value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
				}
			}
			tmp[col]=(uint8_t) (value/4<0 ? 0 : 255);
		}
		else{
			value=0;
			for(wrow=0;wrow<2;wrow++){
				for(wcol=-1;wcol<2;wcol++){
					value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					if(wrow==0){
						value+=window[(wrow)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					}
				}
			}
			tmp[col]=(uint8_t) (value/9<0 ? 0 : 255);
		}
	}
	for(row=1;row<rowcount;row++){
		for(col=0;col<WIDTH;col++){
			if(col==0){
				value=0;
				for(wrow=-1;wrow<2;wrow++){
					for(wcol=0;wcol<2;wcol++){
						value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					}
				}
				temp=tmp[col];
				tmp[col]=(uint8_t) (value/6<0 ? 0 : 255);
			}
			else if(col==WIDTH-1){
				value=0;
				for(wrow=-1;wrow<2;wrow++){
					for(wcol=-1;wcol<1;wcol++){
						value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					}
				}
				frame[framestart+(row-1)*WIDTH+col-1]=temp;
				frame[framestart+(row-1)*WIDTH+col]=tmp[col];
				tmp[col]=(uint8_t) (value/6<0 ? 0 : 255);
			}
			else{
				value=0;
				for(wrow=-1;wrow<2;wrow++){
					for(wcol=-1;wcol<2;wcol++){
						value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					}
				}
				frame[framestart+(row-1)*WIDTH+col-1]=temp;
				temp=tmp[col];
				tmp[col]=(uint8_t) (value/9<0 ? 0 : 255);
			}
		}
	}
	row=rowcount;
	for(col=0;col<WIDTH;col++){ //row equals rowcount
		if(col==0){
			value=0;
			for(wrow=-1;wrow<2;wrow++){
				for(wcol=0;wcol<2;wcol++){
					value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
				}
			}
			temp=tmp[col];
			tmp[col]=(uint8_t) (value/4<0 ? 0 : 255);
		}
		else if(col==WIDTH-1){
			value=0;
			for(wrow=-1;wrow<2;wrow++){
				for(wcol=-1;wcol<1;wcol++){
					value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
				}
			}
			frame[framestart+(row-1)*WIDTH+col-1]=temp;
			frame[framestart+(row-1)*WIDTH+col]=tmp[col];
			tmp[col]=(uint8_t) (value/4<0 ? 0 : 255);
		}
		else{
			value=0;
			for(wrow=-1;wrow<1;wrow++){
				for(wcol=-1;wcol<2;wcol++){
					value+=window[(wrow+1)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					if(wrow==0){
						value+=window[(wrow+2)*3+(wcol+1)]*(int)frame[framestart+row*WIDTH+col+wrow*WIDTH+wcol];
					}
				}
			}
			frame[framestart+(row-1)*WIDTH+col-1]=temp;
			temp=tmp[col];
			tmp[col]=(uint8_t) (value/9<0 ? 0 : 255);
		}
	}
	for(col=0;col<WIDTH;col++){ // Final row written from tmp to frame
		frame[framestart+rowcount*WIDTH+col]=tmp[col];
	}
}

__global__ void filters(uint8_t *frame){ // Function called from host to start filtering
	medianFilter(frame);
	edgeDetection(frame);
}

void* filterThread(void* d_frame){ // Activate devices
	int blocksPerGrid =CORENUM;
	uint8_t* frame=(uint8_t*)d_frame;
	filters<<<blocksPerGrid,1>>>(frame); 
	return 0;
}

/*
*	Kinect Section
*/

void rgb_cb(freenect_device *dev, void *rgb, uint32_t timestamp){
	rgb_display = (uint8_t*)rgb;
}

void *freenect_threadfunc(void *arg){
	freenect_set_tilt_degs(f_dev,0);
	freenect_set_led(f_dev,LED_RED);
	freenect_set_video_callback(f_dev, rgb_cb);
	freenect_set_video_mode(f_dev, freenect_find_video_mode(FREENECT_RESOLUTION_MEDIUM, FREENECT_VIDEO_RGB));
	freenect_start_video(f_dev);
	while (!terminate && (freenect_process_events(f_ctx) >= 0)){}
	printf("\nshutting down streams...\n");
	//freenect_stop_depth(f_dev);
	freenect_stop_video(f_dev);
	freenect_close_device(f_dev);
	freenect_shutdown(f_ctx);
	printf("-- done!\n");
	return NULL;
}

/*
* Host main
*/

int main(int argc, char **argv){
	pthread_t thread;
    int res,count=0;
    hipError_t err = hipSuccess;
	IplImage* grayscale;
    size_t size = SIZE * sizeof(uint8_t);
	rgb_display = (uint8_t*)malloc(size*3);
	gray_display= (uint8_t*)malloc(size);
	tmp= (uint8_t*)malloc(size);
	tmp2= (uint8_t*)malloc(size);
	window=(uint8_t*)malloc(9);
	time_t start,end;
	double seconds,fps;

    uint8_t *d_frame = NULL;
    err = hipMalloc((void **)&d_frame, size);
	
	/* Kinect in main */
		
	printf("Kinect camera test\n");
	if (freenect_init(&f_ctx, NULL) < 0) {
		printf("freenect_init() failed\n");
		return 1;
	}
	//freenect_set_log_level(f_ctx, FREENECT_LOG_DEBUG);
	freenect_select_subdevices(f_ctx, (freenect_device_flags)(FREENECT_DEVICE_MOTOR | FREENECT_DEVICE_CAMERA));
	int nr_devices = freenect_num_devices (f_ctx);
	printf ("Number of devices found: %d\n", nr_devices);
	int user_device_number = 0;
	if (argc > 1)
		user_device_number = atoi(argv[1]);
	if (nr_devices < 1) {
		freenect_shutdown(f_ctx);
		return 1;
	}
	if (freenect_open_device(f_ctx, &f_dev, user_device_number) < 0) {
		printf("Could not open device\n");
		freenect_shutdown(f_ctx);
		return 1;
	}
	res = pthread_create(&freenect_thread, NULL, freenect_threadfunc, NULL);
	if (res) {
		printf("pthread_create failed\n");
		freenect_shutdown(f_ctx);
		return 1;
	}

	/* OpenCV in main */

	cvNamedWindow( "CUDA Example" , CV_WINDOW_AUTOSIZE);
	grayscale = cvCreateImageHeader(cvSize(WIDTH,HEIGHT), IPL_DEPTH_8U, 1);
	time(&start);
	gray(rgb_display);

	/* Start */

	while(1){
		/* HOST */
		//gray(rgb_display);
		//medianFilter(tmp);
		//edgeDetection(tmp2);


		err = hipMemcpy(d_frame, tmp, size, hipMemcpyHostToDevice);
		if (err != hipSuccess){
		    fprintf(stderr, "Failed to copy input frame from host to device (error code %s)!\n", hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		}

		pthread_create( &thread, NULL, filterThread, (void*) d_frame);
		gray(rgb_display);
		pthread_join( thread, NULL);


		err = hipGetLastError();
		if (err != hipSuccess){
		    fprintf(stderr, "Failed to launch filtering kernel (error code %s)!\n", hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		}

		err = hipMemcpy(gray_display, d_frame, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess){
		    fprintf(stderr, "Failed to copy output frame from device to host (error code %s)!\n", hipGetErrorString(err));
		    exit(EXIT_FAILURE);
		}
		cvSetData(grayscale,gray_display,WIDTH);
		cvShowImage("CUDA Example", grayscale);
		++count;
		char c = cvWaitKey(33);
	    if (c == 27){
			time(&end);
			break;
	    }
	}

        seconds=difftime(end, start);
        fps=count/seconds;
        printf("FPS = %.2f\n",fps);
    // Free device global memory
    err = hipFree(d_frame);

    // Free host memory
	free(rgb_display);
	free(gray_display);
	free(tmp);
	free(tmp2);
	free(window);
    err = hipDeviceReset();
	
	// Free kinect
	terminate=1;

	//Free OpenCV window
	cvReleaseImage(&grayscale);
	cvDestroyWindow("CUDA Example");
    return 0;
}
